#include "hip/hip_runtime.h"


__global__ void add(int a, int b, int *c)
{
	*c = a + b;
}

int calculateAddOnGPU(int a, int b)
{
	int* c = new int(0);

	hipMallocManaged(&c, sizeof(int));

	add<<<1, 1>>>(a, b, c);
	hipDeviceSynchronize();

	const unsigned int result = *c;

	hipFree(c);
	hipDeviceReset();

	return result;
}